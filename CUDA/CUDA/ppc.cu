#include "hip/hip_runtime.h"


#include <string>
#include <cstring>
#include <iostream>
#include <cstdio>
#include <vector>
#include <chrono>
#include <map>
#include <fstream>
#include <sstream>

#define limit 150//00000
#define columns 26
#define string_size 255

std::fstream dataset;
std::fstream wdataset;
std::vector<std::string> dictindex;
std::vector<std::vector<std::string>> m_dataset; //matriz que ira armazenar a parte do dataset que devera ser modificado
char h_dataset[limit][12][string_size]; //matriz que ira armazenar a parte do dataset que devera ser modificado
std::string Line; //string responsavel por armazenar a linha que atualmente esta sendo lida
std::fstream dict;
int number = 0;
std::map<int, int> Orietation; //guardar as colunas exatas de onde o dataset sera tratado, para ajudar na escrita final

char h_ids[limit][12][string_size];

std::vector<std::string> blacklist = { "idatracacao", "idcarga", "nacionalidadearmador", "pesocarga_cntr", "pesocargabruta", "qtcarga", "tatracado", "tesperaatracacao", "tesperacainicioop",
"tesperadesatracacao", "testadia", "toperacao", "ano", "cdmercadoria_cntr"}; //os seguintes nomes sao colunas de valores ja numericos, nao precisando de um tratamento de id para tais

bool in_blackList(std::string value) {
    for (int i = 0; i < size(blacklist); i++) {
        if (blacklist[i] == value) {
            return true;
        }
    }

    return false;
}

//Funcao para ler a primeira linha do arquivo e criar os arquivos de dicionarios necessarios
void CreateDictArchive() {
    if (dataset.is_open()) {
        if (std::getline(dataset, Line)) {
            int i = 0;
            wdataset << Line << std::endl; //escrever cabecalho no arquivo
            std::istringstream StringLine(Line);
            std::string data; //armazenar os valores separados por virguma
            while (std::getline(StringLine, data, ',')) { //pegar o valor antes das/entre as virgula e armazenar em data
                if (!in_blackList(data)) {
                    dictindex.push_back("dicts/" + data + ".csv");
                    Orietation[i] = 0;
                    number++;
                    dict.open("dicts/" + data + ".csv", std::ios::out | std::ios::trunc); //excluir conteudo se ja existir
                    dict.close();
                }
                else
                {
                    dictindex.push_back("b");
                }
                i++;
            }
        }
    }
}

//void WriteCsv(int sum) {
//
//    for (int i = 0; i < m_dataset.size(); i++) {
//        Line = m_dataset[i][0];
//        for (int j = 1; j < m_dataset[i].size(); j++) {
//            Line += "," + m_dataset[i][j];
//        }
//        wdataset << Line + '\n';
//    }
//
//}

void WriteCsv(int sum) {
    printf("alo %s", h_dataset[0][0]);
    for (int i = 0; i < m_dataset.size(); i++) {
        wdataset << m_dataset[i][0];
        int index = 1, indext = 0;
        for (int j = 1; j < columns; j++) {
            if (Orietation.find(j) == Orietation.end()) {
                wdataset << "," + m_dataset[i][index];
                index++;
            }
            else {
                wdataset << "," + std::string(h_dataset[i][indext]);
                indext++;
            }
        }
        wdataset << '\n';
    }

}

int Id(int indexc) {
    std::map<std::string, int> ids;
    int indexl = 0;
    bool loop = true;
    while (strlen(h_ids[indexl][indexc]) > 0) {
        ids[h_ids[indexl][indexc]] = indexl + 1;
        indexl++;
    }


    for (int j = 0; j < m_dataset.size(); j++) {
        if (ids.find(m_dataset[j][indexc]) == ids.end()) {
            ids[m_dataset[j][indexc]] = ids.size();
            //strcpy(h_ids[indexl][indexc], "");
            snprintf(h_ids[indexl][indexc], string_size, m_dataset[j][indexc].c_str(), indexl, indexc);
            indexl++;
        }
    }

    ids.clear();

    return indexl;
}


void readCircle() {
    int sum = 0;
    int aux = static_cast<int>((limit * 0.10)); // dividir o servico em 10 partes
    bool can_loop = true;

    while (!dataset.eof() && can_loop || limit >= sum && can_loop) {
        m_dataset.clear();
        for (int i = 0; i < aux; i++) {
            int col = 0; //coluna referencia para o dataset host
            int acol = 0; //coluna atual
            if (std::getline(dataset, Line) && limit >= sum) {
                //separar linha unica em varios componentes para a matriz
                std::istringstream StringLine(Line);
                std::string z_aux;
                std::vector<std::string> V_aux;
                while (std::getline(StringLine, z_aux, ',')) {
                    if (Orietation.find(acol) == Orietation.end()) {
                        V_aux.push_back(z_aux);
                    }
                    else {
                        printf("s");
                        strcpy(h_dataset[i][col], z_aux.c_str());
                        col++;
                    }
                    acol++;
                    
                }
                m_dataset.push_back(V_aux);
                
                //strcy(h_dataset[sum][i])

            }
            else {
                can_loop = false;
            }
            sum++;
        }
        
        //int indexc = -1;
        //for (int i = 0; i < dictindex.size(); i++) {
        //    if (dictindex[i].size() > 1) {
        //        indexc++;
        //        int length = Id(indexc);

        //        std::ofstream dict_out(dictindex[i], std::ios::out | std::ios::trunc); // Excluir conte�do se j� existir
        //        if(dict_out.is_open())
        //        {
        //            dict_out << "Id, Descri��o\n";
        //            for (int j = 0; j < length; j++) {
        //                //std::cout << h_ids[j][i] << " ";
        //                if(strlen(h_ids[j][i]) > 0) dict_out << std::to_string(j + 1) + "," + h_ids[j][i] + "\n";
        //            }
        //            dict_out.close();
        //        }
        //        //printf("%d ",length);
        //    }
        //}

        //datasetManipulation();
        WriteCsv(sum);
    }
}


int main() {
    auto start_time = std::chrono::high_resolution_clock::now();
    wdataset.open("dataset_00_sem_virg_final.csv", std::ios::out | std::ios::trunc);
    //dataset.close();
    dataset.open("dataset_00_sem_virg.csv", std::ios::in | std::ios::out | std::ios::app); //arquivo

    CreateDictArchive();
    //wdataset << '\n';
    readCircle();
    dataset.close();
    wdataset.close();

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    std::cout << "Tempo de execu��o: " << duration << " milissegundos" << std::endl;

    return 0;
}