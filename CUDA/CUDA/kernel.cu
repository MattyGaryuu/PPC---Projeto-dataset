#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <string>
#include <cstring>
#include <iostream>
#include <cstdio>
#include <vector>
#include <chrono>
#include <map>
#include <fstream>
#include <sstream>

#define limit 15000000
#define columns 26
#define string_size 128
#define Max_loop 500000

std::fstream dataset;
std::fstream wdataset;
std::vector<std::string> dictindex;
std::vector<std::vector<std::string>> m_dataset; //matriz que ira armazenar a parte do dataset que devera ser modificado
//char h_dataset[limit][12][string_size]; //matriz que ira armazenar a parte do dataset que devera ser modificado
std::string Line; //string responsavel por armazenar a linha que atualmente esta sendo lida
std::fstream dict;
std::map<int, int> Orietation; //guardar as colunas exatas de onde o dataset sera tratado, para ajudar na escrita final



std::vector<std::string> blacklist = { "idatracacao", "idcarga", "nacionalidadearmador", "pesocarga_cntr", "pesocargabruta", "qtcarga", "tatracado", "tesperaatracacao", "tesperacainicioop",
"tesperadesatracacao", "testadia", "toperacao", "ano", "cdmercadoria_cntr" }; //os seguintes nomes sao colunas de valores ja numericos, nao precisando de um tratamento de id para tais

bool in_blackList(std::string value) {
    for (int i = 0; i < size(blacklist); i++) {
        if (blacklist[i] == value) {
            return true;
        }
    }

    return false;
}

//Funcao para ler a primeira linha do arquivo e criar os arquivos de dicionarios necessarios
void CreateDictArchive() {
    if (dataset.is_open()) {
        if (std::getline(dataset, Line)) {
            int i = 0; //responsavel por saber em qual coluna o token esta
            wdataset << Line << std::endl; //escrever cabecalho no arquivo
            std::istringstream StringLine(Line);
            std::string data; //armazenar os valores separados por virguma
            while (std::getline(StringLine, data, ',')) { //pegar o valor antes das/entre as virgula e armazenar em data
                if (!in_blackList(data)) { //blacklist eh responsavel por armazenar as colunas que nao devem ser tratadas
                    dictindex.push_back("dicts/" + data + ".csv");
                    Orietation[i] = 0; //adicionar coluna exata que devera ser tratada, usar a variavel na escrita do dataset
                    dict.open("dicts/" + data + ".csv", std::ios::out | std::ios::trunc); //excluir conteudo se ja existir
                    dict.close();
                }
                else
                {
                    dictindex.push_back("b"); //poderia ser qualquer valor ate memso vazio
                }
                i++;
            }
        }
    }
}

void Id(int indexc, char h_dataset[Max_loop][12][string_size], char h_ids[3000][12][string_size]) {
    std::map<std::string, int> ids; //mapa para facilitar busca
    int indexl = 0;
    //Varrer a tabela de ids para saber em qual index parou
    for (int i = 0; i < 3000; i++) {
        if(strlen(h_ids[i][indexc]) <= 0) break;
        ids[std::string(h_ids[i][indexc])] = i + 1;
        indexl++;
    }

    for (int j = 0; j < m_dataset.size(); j++) {
        if (ids.find(std::string(h_dataset[j][indexc])) == ids.end()) {
            ids[std::string(h_dataset[j][indexc])] = ids.size();
            snprintf(h_ids[indexl][indexc], string_size, h_dataset[j][indexc], indexl, indexc);
            indexl++;
        }
    }

}

void addDict(char h_ids[3000][12][string_size]) {
    int acol = 0; // variavel para acessar o index exato do char
    for (int i = 0; i < 26; i++) {
        if (dictindex[i].size() > 1) {
            std::ofstream dict_out(dictindex[i], std::ios::out | std::ios::trunc); // Excluir conteúdo se já existir
            if (dict_out.is_open())
            {
                dict_out << "Id, Descrição\n";
                for (int j = 0; j < 3000; j++) {
                    if (strlen(h_ids[j][acol]) > 0) dict_out << std::to_string(j + 1) + "," + h_ids[j][acol] + "\n";
                }
                dict_out.close();
                acol++;
            }
            
        }
    }
}

__global__ void CreateIdDataset(char* d_ids, char* d_dataset, int* d_final_dataset) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // Calcular a linha e a coluna baseado no valor da thread atual
    int row = threadId / 12;
    int col = threadId % 12;

    if (row < Max_loop && col < 12) {
        //Nao ha como comparar strings em device com funcoes como strcmp, entao o grupo para contornar essa situacao, trabalhou com
        //a ideia do que eh uma string na realidade: uma cadeia de char, entao se fez uma funcao para comparar cada caracter individualmente

        char value_to_find[string_size]; 
        for (int i = 0; i < string_size; i++) {
            value_to_find[i] = d_dataset[row * 12 * string_size + col * string_size + i];
        }

        //procurar em ids
        for (int row_ids = 0; row_ids < 3000; row_ids++) {
            bool strings_equal = true; //variavel responsavel para saber se as strings sao iguais

            //comparar
            for (int i = 0; i < string_size; ++i) {
                char id_value = d_ids[row_ids * 12 * string_size + col * string_size + i]; //caracter exato para se comparar

                if (id_value != value_to_find[i]) {
                    strings_equal = false;
                    break; //economizar tempo
                }
            }

            if (strings_equal) {
                d_final_dataset[row *12 + col] = row_ids + 1;

                break;
            }
        }
    }
}

char h_ids[3000][12][string_size]; //matriz de string responsavel por guardar todos os valores, o index da linha eh seu id, por C nao suportar tamanhos gigantescos de matriz
//apos testes usando c++ e openmp, decidimos usar um valor seguro de 3000 linhas
char h_dataset[Max_loop][12][string_size]; //matriz que ira armazenar a parte do dataset que devera ser modificado

void Loop() {
    bool can_loop = true;
    int sum = 0; //controle para saber se limite foi atingido
    int indexl = 0;

    while (!dataset.eof() && can_loop || limit >= sum && can_loop) {
        for (int i = 0; i < Max_loop; i++) {
            int col = 0; //coluna referencia para o dataset host
            int acol = 0; //coluna atual
            if (std::getline(dataset, Line) && limit >= sum) {
                //separar linha unica em varios componentes para a matriz
                std::istringstream StringLine(Line);
                std::string z_aux;
                std::vector<std::string> V_aux;
                while (std::getline(StringLine, z_aux, ',')) {
                    //procurar em orientation para saber se a coluna atual pertence a host
                    if (Orietation.find(acol) == Orietation.end()) {
                        V_aux.push_back(z_aux); //se nao pertencer dar pushback em V_aux
                    }
                    else {
                        strcpy(h_dataset[i][col], z_aux.c_str());
                        col++;
                    }
                    acol++;

                }
                m_dataset.push_back(V_aux);
                sum++;
            }
            else {
                can_loop = false;
            }
            
        }

        //procurar id de cada coluna
        for (int j = 0; j < 12; j++) {
            Id(j, h_dataset, h_ids);
        }

        //tratamento em cuda, o nosso grupo decidiu por fazer uma nova variavel int para que o valor de row seja armazenados nele,
        //ja que funcoes como to string e atoi nao funcionam no device
        char* d_ids;
        char* d_dataset;
        int* d_final_dataset; //responsavel por fazer as modificacoes

        hipMalloc((void**)&d_ids, sizeof(char) * 3000 * 12 * string_size);
        hipMemcpy(d_ids, h_ids, sizeof(char) * 3000 * 12 * string_size, hipMemcpyHostToDevice);

        hipMalloc((void**)&d_dataset, sizeof(char) * Max_loop * 12 * string_size);
        hipMemcpy(d_dataset, h_dataset, sizeof(char) * Max_loop * 12 * string_size, hipMemcpyHostToDevice);

        hipMalloc((void**)&d_final_dataset, sizeof(int) * Max_loop * 12);


        dim3 block_dim(512, 1);
        dim3 grid_dim((Max_loop * 12 + block_dim.x - 1) / block_dim.x, 1);

        // Chame a função do kernel
        CreateIdDataset << <grid_dim, block_dim >> > (d_ids, d_dataset, d_final_dataset);


        hipDeviceSynchronize();

        int* h_final_dataset = new int[Max_loop * 12]; //trazer o dataset final para o host atraves dessa variavel

        hipMemcpy(h_final_dataset, d_final_dataset, sizeof(int) * Max_loop * 12, hipMemcpyDeviceToHost);

        hipFree(d_ids);
        hipFree(d_dataset);
        hipFree(d_final_dataset);

        //escrever no dataset
        for (int i = 0; i < m_dataset.size(); i++) {
            wdataset << m_dataset[i][0];
            int index = 1, indext = 0;
            for (int j = 1; j < 26; j++) {
                if (Orietation.find(j) == Orietation.end()) {
                    
                    wdataset << "," + m_dataset[i][index];
                    index++;
                }
                else {
                        wdataset << "," + std::to_string(h_final_dataset[i * 12 + indext]);
                        indext++;
                }
            }
            wdataset << '\n';

        }

        //resetar memoria
        for (int i = 0; i < Max_loop; ++i) {
            for (int j = 0; j < 12; ++j) {
                memset(h_dataset[i][j], 0, string_size);
            }
        }
        m_dataset.clear(); //limpar m_dataset
        printf("%d ", sum);
    }
}

int main() {
    auto start_time = std::chrono::high_resolution_clock::now();
    wdataset.open("dataset_00_sem_virg_final.csv", std::ios::out | std::ios::trunc);
    //dataset.close();
    dataset.open("dataset_00_sem_virg.csv", std::ios::in | std::ios::out | std::ios::app); //arquivo

    CreateDictArchive();
    Loop();
    addDict(h_ids);
    dataset.close();
    wdataset.close();

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    std::cout << "Tempo de execução: " << duration << " milissegundos" << std::endl;

    return 0;
}