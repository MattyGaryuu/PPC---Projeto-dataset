﻿
#include "hip/hip_runtime.h"

#include <string.h>
#include <stdio.h>

#define LIMIT 150//00000
#define N (1000*1024)


bool in_BlackList(char**, char[], int);

void CreateDictFile(FILE* dataset, FILE* wdataset, char** BlackList, char dict_index[26][201], int sizeBlack) {
    if (dataset != NULL) {
        char Line[500];
        fgets(Line, 500, dataset);
        fprintf(wdataset, "%s\n" , Line); //escrever as colunas no arquivo final
        char* StringLine = strtok(Line, ","); //dividir Linha em tokens, separados por ,
        int i = 0;
        FILE* dict;

        while (StringLine != NULL) { //Enquanto houver tokens
            char* actual = strdup(StringLine); //pegar token atual

            //verificar se o valor esta na blacklist
            if (in_BlackList(BlackList, actual, sizeBlack)) {
                strcpy(dict_index[i], "b");
            }
            else {
                snprintf(dict_index[i], 201, "dicts/%s.csv", actual);
                dict = fopen(dict_index[i], "wb");
                if (dict != NULL) {
                    fprintf(dict, "id, descrição");
                    //funcao para preencher id
                    fclose(dict);
                }
            }
            StringLine = strtok(NULL, ","); //Proximo token
            i++;
        }

    }
}

bool in_BlackList(char** BackList, char Value[], int size) {

    for (int i = 0; i < size; i++) {
        if (strcmp(BackList[i], Value) == 0) {
            return true;
        }
    }
    return false;
}

void readCircle() {

}

int main()
{
    FILE* dataset = fopen("dataset_00_sem_virg.csv", "r");
    FILE* wdataset = fopen("dataset_00_sem_virg_final.csv", "w");

    char* Blacklist[] = { "idatracacao", "idcarga", "nacionalidadearmador", "pesocarga_cntr", "pesocargabruta", "qtcarga", "tatracado", "tesperaatracacao", "tesperacainicioop",
"tesperadesatracacao", "testadia", "toperacao", "ano" }; //os seguintes nomes sao colunas de valores ja numericos, nao precisando de um tratamento de id para tais
    char dict_index[26][201];

    CreateDictFile(dataset,wdataset,Blacklist,dict_index, sizeof(Blacklist) / sizeof(Blacklist[0]));

    return 0;
}
