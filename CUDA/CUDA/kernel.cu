
#include <hip/hip_runtime.h>
﻿//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include <string.h>
//#include <stdio.h>
//#include <time.h>
//
//#define LIMIT 150//00000
//#define N (1000*1024)
//
//
//bool in_BlackList(char**, char[], int);
//
//void CreateDictFile(FILE* dataset, FILE* wdataset, char** BlackList, char dict_index[26][201], int sizeBlack) {
//    if (dataset != NULL) {
//        char Line[500];
//        fgets(Line, 500, dataset);
//        fprintf(wdataset, "%s\n" , Line); //escrever as colunas no arquivo final
//        char* StringLine = strtok(Line, ","); //dividir Linha em tokens, separados por ,
//        int i = 0;
//        FILE* dict;
//
//        while (StringLine != NULL) { //Enquanto houver tokens
//            char* actual = strdup(StringLine); //pegar token atual
//
//            //verificar se o valor esta na blacklist
//            if (in_BlackList(BlackList, actual, sizeBlack)) {
//                strcpy(dict_index[i], "b");
//            }
//            else {
//                snprintf(dict_index[i], 201, "dicts/%s.csv", actual);
//                dict = fopen(dict_index[i], "wb");
//                if (dict != NULL) {
//                    fprintf(dict, "id, descrição");
//
//                    //funcao para preencher id
//                    fclose(dict);
//                }
//            }
//            StringLine = strtok(NULL, ","); //Proximo token
//            i++;
//        }
//
//    }
//}
//
//bool in_BlackList(char** BackList, char Value[], int size) {
//
//    for (int i = 0; i < size; i++) {
//        if (strcmp(BackList[i], Value) == 0) {
//            return true;
//        }
//    }
//    return false;
//}
//
//void datasetManipulation(char dict_index[26][201],int d_aux, char m_dataset[(int)(LIMIT * 0.10) + 1][26][201]) {
//    char Line[500];
//    char ids[LIMIT][201];
//    
//    for (int i = 0; i < 26; i++) {
//        if(strlen(dict_index[i]) > 1) {
//            int index = 0;
//            FILE* dict = fopen(dict_index[i], "r");
//            if (dict != NULL) {
//                fgets(Line, 500, dict); //ler primeira linha para ignorar
//                while (fgets(Line, 500, dict)) {
//                    char* StringLine = strtok(Line, ",");
//                    StringLine = strtok(NULL, ","); //ignorar primeira id
//                    strcpy(ids[index], strdup(StringLine));
//                    index++;
//                }
//                fclose(dict);
//            }
//
//
//            //preencher ids
//            for (int j = d_aux+1; j > 0; j--) {
//                bool in_id = false;
//                for (int z = 0; z < index; z++) {
//                    if (strcmp(ids[z], m_dataset[j][i]) == 0) {
//                        in_id = true;
//                    }
//                }
//                if (!in_id) {
//                    strcpy(ids[index], m_dataset[j][i]);
//                    printf("%s", ids[index]);
//                    index++;
//                }
//            }
//
//            dict = fopen(dict_index[i], "w");
//            if (dict != NULL) {
//                fprintf(dict, "id, descrição\n");
//                for (int j = 0; j < index; j++) {
//                    fprintf(dict, "%d,%s\n", (j + 1), ids[j]);
//                }
//                fclose(dict);
//            }
//        }
//    }
//}
//
//void readCircle(FILE* dataset, char dict_index[26][201]) {
//    int aux = (int)(LIMIT * 0.10) + 1;
//    int sum = 1;
//    char Line[500];
//    bool can_loop = true;
//    fgets(Line, sizeof(Line), dataset);
//    
//    while(LIMIT >= sum)
//    {
//        int d_aux = 0;
//        char m_dataset[(int)(LIMIT * 0.10) + 1][26][201];
//        while (fgets(Line, sizeof(Line), dataset) != NULL && aux > d_aux) {
//            //printf("%s\n", Line);
//            char* StringLine = strtok(Line, ",");
//            int i = 0;
//            while (StringLine != NULL) { //Enquanto houver tokens
//                char* actual = strdup(StringLine); //pegar token atual
//                //printf("%s\n", actual);
//                if (strlen(actual) > 0) {
//                    strcpy(m_dataset[d_aux][i], actual);
//                }
//                //printf(" %s ", m_dataset[d_aux][i]);
//                StringLine = strtok(NULL, ",");
//                i++;
//            }
//            //printf("%s ", m_dataset[][0]);
//            //datasetManipulation(dict_index,d_aux, m_dataset);
//            
//            sum++;
//            d_aux++;
//        }
//for (int z = 0; z < (int)(LIMIT * 0.10) + 1; z++) {
//                printf("linha %d: ", z);
//                for (int j = 0; j < 26; j++) {
//                    //printf("%s ", m_dataset[z][j]);
//                }
//                printf("\n");
//            }
//
//        
//    }
//    
//}
//
//int main()
//{
//
//    FILE* dataset = fopen("dataset_00_sem_virg.csv", "r");
//    FILE* wdataset = fopen("dataset_00_sem_virg_final.csv", "w");
//
//    char* Blacklist[] = { "idatracacao", "idcarga", "nacionalidadearmador", "pesocarga_cntr", "pesocargabruta", "qtcarga", "tatracado", "tesperaatracacao", "tesperacainicioop",
//"tesperadesatracacao", "testadia", "toperacao", "ano" }; //os seguintes nomes sao colunas de valores ja numericos, nao precisando de um tratamento de id para tais
//    char dict_index[26][201];
//
//    CreateDictFile(dataset,wdataset,Blacklist,dict_index, sizeof(Blacklist) / sizeof(Blacklist[0]));
//    readCircle(dataset, dict_index);
//    return 0;
//}
